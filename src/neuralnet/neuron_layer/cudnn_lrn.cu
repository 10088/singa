/************************************************************
*
* Licensed to the Apache Software Foundation (ASF) under one
* or more contributor license agreements.  See the NOTICE file
* distributed with this work for additional information
* regarding copyright ownership.  The ASF licenses this file
* to you under the Apache License, Version 2.0 (the
* "License"); you may not use this file except in compliance
* with the License.  You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing,
* software distributed under the License is distributed on an
* "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
* KIND, either express or implied.  See the License for the
* specific language governing permissions and limitations
* under the License.
*
*************************************************************/

#include "singa/neuralnet/neuron_layer.h"

namespace singa {
CudnnLRNLayer::~CudnnLRNLayer() {
  if (!init_cudnn_) {
    hipdnnDestroyLRNDescriptor(norm_desc_);
  }
}

void CudnnLRNLayer::Setup(const LayerProto& proto,
    const vector<Layer*>& srclayers) {
  LRNLayer::Setup(proto, srclayers);
  mode_ = HIPDNN_LRN_CROSS_CHANNEL;
}

void CudnnLRNLayer::InitCudnn() {
  CudnnLayer::InitCudnn(srclayers);
  CHECK_EQ(hipdnnCreateLRNDescriptor(&norm_desc_), HIPDNN_STATUS_SUCCESS);
  CHECK_EQ(hipdnnSetLRNDescriptor(norm_desc_,
        lsize_,
        alpha_,
        beta_,
        knorm_), HIPDNN_STATUS_SUCCESS);
  CHECK_EQ(hipdnnCreateTensorDescriptor(&src_desc_), HIPDNN_STATUS_SUCCESS);
  CHECK_EQ(hipdnnSetTensor4dDescriptor(src_desc_,
      HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT,
      batchsize_,
      channels_,
      height_,
      width_), HIPDNN_STATUS_SUCCESS);
  CHECK_EQ(hipdnnCreateTensorDescriptor(&my_desc_), HIPDNN_STATUS_SUCCESS);
  CHECK_EQ(hipdnnSetTensor4dDescriptor(my_desc_,
      HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT,
      batchsize_,
      channels_,
      height_,
      width_), HIPDNN_STATUS_SUCCESS);
}
void ComputeFeature(int flag, const vector<Layer*>& srclayers) {
  if (init_cudnn_) {
    InitCudnn();
    init_cudnn_ = false;
  }
  CHECK_EQ(hipdnnLRNCrossChannelForward(handle_,
      norm_desc_,
      mode_,
      &alpha,
      src_desc_,
      srclayers[0]->data(this).gpu_data(),
      &beta,
      my_desc_,
      data_.mutable_gpu_data()), HIPDNN_STATUS_SUCCESS);
}
void ComputeGradient(int flag, const vector<Layer*>& srclayers) {
  CHECK_EQ(hipdnnLRNCrossChannelBackward(handle_,
        norm_desc_,
        mode_,
        &alpha,
        my_desc_, // ???
        data_.gpu_data(),
        my_desc_,
        grad_.gpu_data()
        src_desc_,
        srclayers[0]->data(this).gpu_data(),
        &beta,
        src_desc_,
        srclayers[0]->mutable_grad(this)->mutable_gpu_data()),
      HIPDNN_STATUS_SUCCESS);
}


} /* singa */
